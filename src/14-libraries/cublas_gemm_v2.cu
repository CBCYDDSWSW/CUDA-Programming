#include <stdio.h>
#include"error.cuh"
#include<hipblas.h>

void print_matrix(int R, int C, double* A, const char* name) {
    printf("%s:\n", name);
    for (int i = 0; i < R; i++) {
        for (int j = 0; j < C; j++) {
            printf("%10.6f", A[i * C + j]);
        }
        printf("\n");
    }
}


int main() {
    int M = 2, K = 3, N = 4;

    int MK = M * K;
    int KN = K * N;
    int MN = M * N;

    double* h_A = (double*)malloc(MK * sizeof(double));
    double* h_B = (double*)malloc(KN * sizeof(double));
    double* h_C = (double*)malloc(MN * sizeof(double));

    for (int i = 0; i < MK; i++) {
        h_A[i] = i;
    }
    print_matrix(M, K, h_A, "A");

    for (int i = 0; i < KN; i++) {
        h_B[i] = i;
    }
    print_matrix(K, N, h_B, "B");

    for (int i = 0; i < MN; i++) {
        h_C[i] = 0;
    }

    double* g_A, * g_B, * g_C;
    CHECK(hipMalloc((void**)&g_A, sizeof(double) * MK));
    CHECK(hipMalloc((void**)&g_B, sizeof(double) * KN));
    CHECK(hipMalloc((void**)&g_C, sizeof(double) * MN));

    hipblasSetVector(MK, sizeof(double), h_A, 1, g_A, 1);
    hipblasSetVector(KN, sizeof(double), h_B, 1, g_B, 1);
    hipblasSetVector(MN, sizeof(double), h_C, 1, g_C, 1);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    double alpha = 1.0, beta = 0.0;
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, g_A, M, g_B, K, &beta, g_C, M);
    hipblasDestroy(handle);

    hipblasGetVector(MN, sizeof(double), g_C, 1, h_C, 1);
    print_matrix(M, N, h_C, "C");

    free(h_A);
    free(h_B);
    free(h_C);

    CHECK(hipFree(g_A));
    CHECK(hipFree(g_B));
    CHECK(hipFree(g_C));

    return 0;


}



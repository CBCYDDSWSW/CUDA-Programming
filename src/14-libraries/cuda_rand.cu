
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<hiprand/hiprand.h>

void output_result(double* y) {
    FILE* fid = fopen("x1.txt", "w");
    for (int i = 0; i < 100000; i++) {
        fprintf(fid, "%lf\n", y[i]);
    }
    fclose(fid);
}

int main() {
    hiprandGenerator_t generator;
    hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(generator, 1234);
    int N = 100000;
    double* x;
    hipMalloc((void**)&x, N * sizeof(double));
    hiprandGenerateUniformDouble(generator, x, N);

    double* y = (double*)malloc(N *sizeof(double));
    hipMemcpy(y, x, sizeof(double) * N, hipMemcpyDeviceToHost);

    output_result(y);
    hipFree(x);
    free(y);
    hiprandDestroyGenerator(generator);
    return 0;
}